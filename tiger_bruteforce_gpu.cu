#include "hip/hip_runtime.h"
// tiger_bruteforce_gpu.cu
#include <stdio.h>
#include <string.h>
#include <time.h>
#include "tiger_gpu.h"
#include <hip/hip_runtime.h>
#include "tiger.h"
#include "tiger_common.h"

// Constants for GPU bruteforce
#define THREADS_PER_BLOCK 256
#define NUM_BLOCKS 1024
#define CHARSET_SIZE 62 // a-z, A-Z, 0-9

__constant__ char d_charset[CHARSET_SIZE];
__constant__ unsigned char d_target[24];

// Helper function to generate test strings
// Modified portion of tiger_bruteforce_gpu.cu
__device__ void generate_string(char *buffer, size_t length, uint64_t index)
{
    for (size_t i = 0; i < length; i++)
    {
        buffer[i] = d_charset[index % CHARSET_SIZE];
        index /= CHARSET_SIZE;
    }
    buffer[length] = '\0';
}

// Atomic add for 64-bit integers
__device__ unsigned long long atomicAdd64(unsigned long long *address, unsigned long long val)
{
    unsigned long long old = *address, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address, assumed, val + assumed);
    } while (assumed != old);
    return old;
}

// Bruteforce kernel
__global__ void bruteforce_kernel(size_t length, uint64_t start_index, bool *found,
                                  char *result_string, unsigned long long *attempts)
{
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;
    uint64_t current_index = start_index + tid;

    char test_string[32]; // Max length we'll test
    unsigned char hash[24];
    GPU_TIGER_CTX context;

    while (!(*found))
    {
        // Generate test string from current_index
        generate_string(test_string, length, current_index);

        // Compute hash
        TIGERInit_gpu(&context);
        TIGERUpdate_gpu(&context, (const unsigned char *)test_string, length);
        TIGER192Final_gpu(hash, &context);

        atomicAdd64(attempts, 1ULL);

        // Compare hash with target
        bool match = true;
        for (int i = 0; i < 24; i++)
        {
            if (hash[i] != d_target[i])
            {
                match = false;
                break;
            }
        }

        if (match)
        {
            *found = true;
            // Copy the found string to result
            for (size_t i = 0; i <= length; i++)
            {
                result_string[i] = test_string[i];
            }
            return;
        }

        current_index += stride;
    }
}

bool bruteforce_gpu(const unsigned char *target_hash, size_t length, double time_limit,
                    char *result, uint64_t *total_attempts)
{
    bool *d_found;
    char *d_result;
    unsigned long long *d_attempts;
    bool h_found = false;
    unsigned long long h_attempts = 0;
    hipError_t err;

    // Initialize CUDA memory
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_target), target_hash, 24);
    checkCudaError(err, "Failed to copy target hash to constant memory");

    err = hipMalloc(&d_found, sizeof(bool));
    checkCudaError(err, "Failed to allocate device memory for found flag");

    err = hipMalloc(&d_result, 32); // Max string length + null terminator
    checkCudaError(err, "Failed to allocate device memory for result");

    err = hipMalloc(&d_attempts, sizeof(unsigned long long));
    checkCudaError(err, "Failed to allocate device memory for attempts counter");

    err = hipMemset(d_found, 0, sizeof(bool));
    checkCudaError(err, "Failed to initialize found flag");

    err = hipMemset(d_attempts, 0, sizeof(unsigned long long));
    checkCudaError(err, "Failed to initialize attempts counter");

    uint64_t start_index = 0;
    clock_t start_time = clock();

    while ((double)(clock() - start_time) / CLOCKS_PER_SEC < time_limit && !h_found)
    {
        bruteforce_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
            length, start_index, d_found, d_result, d_attempts);

        err = hipGetLastError();
        checkCudaError(err, "Failed to launch bruteforce kernel");

        err = hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
        checkCudaError(err, "Failed to copy found flag from device");

        start_index += NUM_BLOCKS * THREADS_PER_BLOCK;

        // Periodically update attempts count
        if (start_index % (NUM_BLOCKS * THREADS_PER_BLOCK * 100) == 0)
        {
            err = hipMemcpy(&h_attempts, d_attempts, sizeof(unsigned long long), hipMemcpyDeviceToHost);
            checkCudaError(err, "Failed to copy attempts counter from device");
            *total_attempts = (uint64_t)h_attempts;
        }
    }

    // Get final attempt count
    err = hipMemcpy(&h_attempts, d_attempts, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    checkCudaError(err, "Failed to copy final attempts counter from device");
    *total_attempts = (uint64_t)h_attempts;

    // If found, copy the result string
    if (h_found)
    {
        err = hipMemcpy(result, d_result, length + 1, hipMemcpyDeviceToHost);
        checkCudaError(err, "Failed to copy result string from device");
    }

    // Cleanup
    hipFree(d_found);
    hipFree(d_result);
    hipFree(d_attempts);

    return h_found;
}

int main()
{
    initialize_gpu_tables();

    // Test parameters
    const size_t max_length = 8;    // Maximum string length to test
    const double time_limit = 10.0; // Time limit per length in seconds

    printf("Starting GPU bruteforce test\n");
    printf("Testing strings up to length %zu\n", max_length);
    printf("Time limit per length: %.1f seconds\n\n", time_limit);

    for (size_t length = 1; length <= max_length; length++)
    {
        // Create a random target string and its hash
        char target_string[32];
        unsigned char target_hash[24];
        TIGER_CTX context;

        // Generate random target string
        for (size_t i = 0; i < length; i++)
        {
            target_string[i] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"[rand() % 62];
        }
        target_string[length] = '\0';

        // Generate its hash
        TIGERInit(&context);
        TIGERUpdate(&context, (const unsigned char *)target_string, length);
        TIGER192Final(target_hash, &context);

        printf("\nTesting length %zu\n", length);
        printf("Target string: %s\n", target_string);
        printf("Target hash: ");
        for (int i = 0; i < 24; i++)
            printf("%02x", target_hash[i]);
        printf("\n");

        // Try to find it
        char result[32];
        uint64_t attempts = 0;
        bool found = bruteforce_gpu(target_hash, length, time_limit, result, &attempts);

        if (found)
        {
            printf("Found match: %s\n", result);
            printf("Attempts: %lu\n", attempts);
            printf("Speed: %.2f million hashes/second\n",
                   (attempts / time_limit) / 1000000.0);
        }
        else
        {
            printf("No match found within time limit\n");
            printf("Attempts: %lu\n", attempts);
            printf("Speed: %.2f million hashes/second\n",
                   (attempts / time_limit) / 1000000.0);
            break; // Stop if we can't find a match at this length
        }
    }

    return 0;
}